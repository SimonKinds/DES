#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include "string.h"

#include "des.h"
#include "constants.h"

void permutate(const uint8_t* input_array,
    const unsigned int input_bit_count,
    uint8_t* output_array,
    const unsigned int output_bit_cunt,
    const unsigned int* permutation_array) {

  for(unsigned int i = 0; i < output_bit_cunt; ++i) {
    // all indexes in the permutation arrays are starting at 1
    unsigned int original_pos = permutation_array[i] - 1;

    // starts counting from left to right (MSB = 0)
    uint8_t original_index = (input_bit_count - original_pos - 1) / 8;
    uint8_t original_bit_pos = (input_bit_count - original_pos + 7) % 8;

    // use a bit mask to only have it be one bit, in the LSB
    uint8_t original_value = (input_array[original_index] & ( 1 << original_bit_pos )) >> original_bit_pos;

    uint8_t new_index = (output_bit_cunt - i - 1) / 8;
    uint8_t new_bit_pos = (output_bit_cunt - i + 7) % 8;
    output_array[new_index] |= original_value << new_bit_pos;
  }
}

__device__
void permutate_gpu(const uint8_t* input_array,
    const unsigned int input_bit_count,
    uint8_t* output_array,
    const unsigned int output_bit_cunt,
    const unsigned int* permutation_array) {
  for(unsigned int i = 0; i < output_bit_cunt; ++i) {
    // all indexes in the permutation arrays are starting at 1
    unsigned int original_pos = permutation_array[i] - 1;

    // starts counting from left to right (MSB = 0)
    uint8_t original_index = (input_bit_count - original_pos - 1) / 8;
    uint8_t original_bit_pos = (input_bit_count - original_pos + 7) % 8;

    // use a bit mask to only have it be one bit, in the LSB
    uint8_t original_value = (input_array[original_index] & ( 1 << original_bit_pos )) >> original_bit_pos;

    uint8_t new_index = (output_bit_cunt - i - 1) / 8;
    uint8_t new_bit_pos = (output_bit_cunt - i + 7) % 8;
    output_array[new_index] |= original_value << new_bit_pos;
  }
}

__device__
uint64_t initial_permutation(const uint64_t* message) {
  uint64_t permutated = 0;
  permutate_gpu((uint8_t*)message, BLOCK_SIZE, (uint8_t*)&permutated, BLOCK_SIZE, IP_PERMUTATION_ARRAY);
  return permutated;
}

__device__
uint64_t inverse_initial_permutation(const uint64_t* data) {
  uint64_t permutated = 0;
  permutate_gpu((uint8_t*)data, BLOCK_SIZE, (uint8_t*)&permutated, BLOCK_SIZE, INVERSE_IP_PERMUTATION_ARRAY);
  return permutated;
}

uint64_t key_permutation_first(const uint64_t* key) {
  uint64_t permutated = 0;
  permutate((uint8_t*)key, BLOCK_SIZE, (uint8_t*)&permutated, KEY_SIZE_FIRST_PERMUTATION, PC_1);
  return permutated;
}

uint64_t key_permutation_second(const uint64_t* key) {
  uint64_t permutated = 0;
  permutate((uint8_t*)key, KEY_SIZE_FIRST_PERMUTATION, (uint8_t*)&permutated, KEY_SIZE_SECOND_PERMUTATION, PC_2);
  return permutated;
}

__device__
uint64_t expansion_permutation(const uint32_t* r) {
  uint64_t permutated = 0;
  permutate_gpu((uint8_t*)r, BLOCK_SIZE / 2, (uint8_t*)&permutated, KEY_SIZE_SECOND_PERMUTATION, E);
  return permutated;
}

__device__
uint32_t p_permutation(const uint32_t* data) {
  uint32_t permutated = 0;
  permutate_gpu((uint8_t*)data, BLOCK_SIZE / 2, (uint8_t*)&permutated, BLOCK_SIZE / 2, P);
  return permutated;
}

uint32_t left_shift_rotate(const uint32_t data, const unsigned int shifts) {
  uint32_t x = data << shifts;

  // rotate the bits that got overflowed from the 28 LSB
  x |= x >> 28;
  // only use the 28 LSB
  x &= 0x0FFFFFFF;

  return x;
}

uint64_t* generate_subkeys(const uint64_t key) {
  // 56 bits
  uint64_t permutated_key = key_permutation_first(&key);

  const uint32_t c0 = permutated_key >> KEY_SIZE_FIRST_PERMUTATION / 2;
  const uint32_t d0 = (permutated_key << KEY_SIZE_FIRST_PERMUTATION / 2) >> KEY_SIZE_FIRST_PERMUTATION / 2;

  uint32_t cs[AMOUNT_OF_KEYS];
  uint32_t ds[AMOUNT_OF_KEYS];

  cs[0] = left_shift_rotate(c0, KEY_SHIFT_ARRAY[0]);
  ds[0] = left_shift_rotate(d0, KEY_SHIFT_ARRAY[0]);

  for(unsigned int i = 1; i < AMOUNT_OF_KEYS; ++i) {
    cs[i] = left_shift_rotate(cs[i - 1], KEY_SHIFT_ARRAY[i]);
    ds[i] = left_shift_rotate(ds[i - 1], KEY_SHIFT_ARRAY[i]);
  }

  uint64_t* subkeys = (uint64_t*)malloc(AMOUNT_OF_KEYS * sizeof(uint64_t));
  for(unsigned int i = 0; i < AMOUNT_OF_KEYS; ++i) {
    uint64_t k_pre_permutation = concatCD(cs[i], ds[i]);
    uint64_t k = key_permutation_second(&k_pre_permutation);

    subkeys[i] = k;
  }

  return subkeys;
}

uint64_t* reverse_order(const uint64_t* subkeys) {
  uint64_t* reversed = (uint64_t*)malloc(AMOUNT_OF_KEYS * sizeof(uint64_t));

  for(unsigned int i = 0; i < AMOUNT_OF_KEYS; ++i) {
    reversed[ AMOUNT_OF_KEYS - i - 1 ] = subkeys[i];
  }

  return reversed;
}

// the permutated key is 56 bits, and we want the 28 MSB
uint32_t getC0(const uint64_t* permutated_key) {
  // 3x8 = 24
  // 4 bits are from the LSB
  // we now have the 24 LSB of the MSB of the permutated key
  unsigned int start_index = 3;
  uint32_t c0 = (permutated_key[start_index] & 0xF0) >> 4;

  for(unsigned int i = 1; i < KEY_SIZE_FIRST_PERMUTATION / 8 - start_index; ++i) {
    c0 |= permutated_key[i + start_index] << (4 + (8 * (i - 1)));
  }

  return c0;
}

uint32_t getD0(const uint8_t* permutated_key) {
  uint32_t d0 = *((uint32_t*) permutated_key) & 0x0FFFFFFF;

  return d0;
}

uint64_t concatCD(const uint32_t c, const uint32_t d) {
  uint64_t concat = c;
  concat <<= 28;
  concat |= d;

  return concat;
}

__device__
uint32_t feistal(const uint32_t* r, const uint64_t* key) {
  const uint64_t expanded = expansion_permutation(r);
  const uint64_t xored = expanded ^ (*key);

  uint32_t s_transformed = s_box_transformation(&xored);

  const uint32_t p_permutated = p_permutation(&s_transformed);

  return p_permutated;
}

//  S8 is LSB, S1 is MSB
// data is 48 bits
__device__
uint32_t s_box_transformation(const uint64_t* data) {

  uint32_t val = 0;
  // use 6 LSB
  val = s_value((*data << 42) >> 42, S8_BOX);
  // use 4 LSB from *data as MSB and 2 MSB from *data as LSB
  val |= s_value((*data << 36) >> 42, S7_BOX)  << 4;
  // use 2 LSB from *data as MSB and 4 MSB from *data as LSB
  val |= s_value((*data << 30) >> 42, S6_BOX) << 8;
  // use 6 MSB
  val |= s_value((*data << 24) >> 42, S5_BOX) << 12;

  // use 6 LSB
  val |= s_value((*data << 18) >> 42, S4_BOX) << 16;
  // use 4 LSB from *data as MSB and 2 MSB from *data as LSB
  val |= s_value((*data << 12) >> 42, S3_BOX) << 20;
  // use 2 LSB from *data as MSB and 4 MSB from *data as LSB
  val |= s_value((*data << 6) >> 42, S2_BOX) << 24;
  // use 6 MSB
  val |= s_value(*data >> 42, S1_BOX) << 28;

  return val;
}

__device__
uint8_t s_value(const uint8_t b, const unsigned int* s_box) {
  // first and last
  unsigned int i = ((b & 0x20) >> 4) | (b & 0x1);
  // middle 4
  unsigned int j = (b & 0x1E) >> 1;

  return s_box[i * 16 + j];
}

__device__
uint32_t calculate_r(const uint32_t prev_l, const uint32_t prev_r, const uint64_t* key) {
  return prev_l ^ feistal(&prev_r, key);
}

__global__
void des(const uint64_t* message, const uint64_t* subkeys, uint64_t* output_block) {
  const uint64_t permutated = initial_permutation(message);

  uint32_t l = permutated >> 32;

  // will only use the 32 LSB
  uint32_t r = (permutated << 32) >> 32;

  for(unsigned int i = 0; i < AMOUNT_OF_KEYS; ++i) {
    uint32_t prev_l = l;
    l = r;
    r = calculate_r(prev_l, r, &subkeys[i]);
  }

  uint64_t concat = r;
  concat <<= 32;
  concat |= l;

  *output_block = inverse_initial_permutation(&concat);
}

uint64_t* encode(const uint64_t* message, const unsigned int size, const uint64_t key) {

  uint64_t* message_gpu;
  hipMalloc(&message_gpu, size);
  hipMemcpy(message_gpu, message, size, hipMemcpyHostToDevice);

  uint64_t* subkeys = generate_subkeys(key);
  uint64_t* subkeys_gpu;
  hipMalloc(&subkeys_gpu, 16 * sizeof(uint64_t));
  hipMemcpy(subkeys_gpu, subkeys, 16 * sizeof(uint64_t), hipMemcpyHostToDevice);

  free(subkeys);


  uint64_t* encoded_message_gpu;
  hipMalloc(&encoded_message_gpu, size);

  for(unsigned int i = 0; i < size / sizeof(uint64_t); ++i) {
    des<<< 1,1 >>>(&message_gpu[i], subkeys_gpu, &encoded_message_gpu[i]);
  }

  uint64_t* encoded_message = (uint64_t*) malloc(size);
  hipMemcpy(encoded_message, encoded_message_gpu, size, hipMemcpyDeviceToHost);

  hipFree(subkeys_gpu);
  hipFree(message_gpu);
  hipFree(encoded_message_gpu);

  return encoded_message;
}

uint64_t* decode(const uint64_t* encoded, const unsigned int size, const uint64_t key) {
  uint64_t* subkeys = generate_subkeys(key);
  uint64_t* reversed_subkeys = reverse_order(subkeys);

  uint64_t* decoded_message;
  hipMalloc(&decoded_message, size);

  for(unsigned int i = 0; i < size / sizeof(uint64_t); ++i) {
     des<<< 1,1 >>>(&encoded[i], reversed_subkeys, &decoded_message[i]);
  }

  free(subkeys);
  free(reversed_subkeys);
  return decoded_message;
}

uint64_t pkcs5_padding(const uint64_t* block, unsigned int amount_of_bytes_to_pad) {
  uint64_t padded = *block;
  padded <<= amount_of_bytes_to_pad * 8;
  for(unsigned int i = 0; i < amount_of_bytes_to_pad; ++i) {
    ((uint8_t*)&padded)[i] = amount_of_bytes_to_pad;
  }

  return padded;
}

unsigned int count_padding_bytes(const uint64_t* block) {
  const uint8_t* bytes = (const uint8_t*) block;
  const unsigned int amount_of_bytes = BLOCK_SIZE / 8;; 
  const uint8_t prev_value = bytes[0];
  for(unsigned int i = 1; i < amount_of_bytes; ++i) {
    //if they're no longer equal, it should be the end of the padding
    if(bytes[i] != prev_value) {
      //but if it's not equal to the amount of padding added, then was not actually padding
      if(prev_value == i) {
        return i;
      }

      //if they were not equal and it was not equal to i, it was not padding
      break;
    }
  }

  return 0;
}

void print_help_message() {
  printf("Usage:\n\
      des -e input -k key output\n\
      des -d input -k key output\n");
}

void print_key_error_msg() {
  printf("Key has to be 8 characters long\n");
}

void write_to_file(const char* file_name, const uint8_t* data, const unsigned int file_size) {
  FILE* output_file = fopen(file_name, "wb");
  if(output_file == NULL) {
    printf("Could not open output file named %s", file_name);
  }
  //the amount of elements read is also the amounts of elements to be written
  const size_t elements_written = fwrite(data, sizeof(uint8_t), file_size, output_file);
  if(elements_written != file_size) {
    printf("Could not write to output file");
  }
  fclose(output_file);
}

int main(int argc, char** argv) {

  if(argc != 6) {
    print_help_message();
    return 0;
  }
  const char* input_file_name = argv[2];
  const char* key_string = argv[4];
  if(strlen(key_string) != 8) {
    print_key_error_msg();
    return 1;
  }
  const uint64_t key = *(uint64_t*)key_string;
  const char* output_file_name = argv[5];
  FILE* input_file = fopen(input_file_name, "rb");

  // obtain file size:
  fseek(input_file , 0 , SEEK_END);
  // in bytes
  const uint64_t file_size = ftell(input_file);
  rewind(input_file);

  //ceil the amount of elements to be read
  const size_t elements_to_read = (file_size + sizeof(uint64_t) - 1) / sizeof(uint64_t);

  uint64_t* file_buffer = (uint64_t*)malloc(elements_to_read * sizeof(uint64_t));
  fread(file_buffer, sizeof(uint8_t), file_size, input_file);

  fclose(input_file);

  if(strcmp(argv[1], "-e") == 0) {
    // add padding to the last element (if needed)
    const unsigned int output_byte_count = elements_to_read * sizeof(uint64_t);
    file_buffer[elements_to_read - 1] = pkcs5_padding(&file_buffer[elements_to_read - 1], output_byte_count - file_size);

    uint64_t* encoded_file_buffer = encode(file_buffer, output_byte_count, key);

    write_to_file(output_file_name, (uint8_t*)encoded_file_buffer, output_byte_count);

    free(encoded_file_buffer);

  } else if(strcmp(argv[1], "-d") == 0) {
    uint64_t* decoded_file_buffer = decode(file_buffer, file_size, key);
    const size_t bytes_of_padding = count_padding_bytes(decoded_file_buffer);
    decoded_file_buffer[ file_size / sizeof(uint64_t) - 1 ] >>= bytes_of_padding * 8;

    const size_t output_byte_count = file_size - bytes_of_padding;

    write_to_file(output_file_name, (uint8_t*)decoded_file_buffer, output_byte_count);

    free(decoded_file_buffer);
  } else {
    print_help_message();
  }

  free(file_buffer);
  return 0;
}
